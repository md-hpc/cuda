#include "hip/hip_runtime.h"
extern "C" {

#include "pdb_importer.h"

}
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <assert.h>

#define MAX_PARTICLES_PER_BLOCK 1024
//#define EPSILON (1.65e-9)                       // ng * m^2 / s^2
#define EPSILON (1.65e11)                        // ng * A^2 / s^2
#define ARGON_MASS (39.948 * 1.66054e-15)       // ng
#define SIGMA (0.034f)                           // A
#define GPU_PERROR(err) do {\
    if (err != hipSuccess) {\
        fprintf(stderr,"gpu_perror: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);\
        exit(err);\
    }\
} while (0);

// constexpr float LJMAX = (4.0f * 24.0f * EPSILON / SIGMA * (powf(7.0f / 26.0f, 7.0f / 6.0f) - 2.0f * powf(7.0f / 26.0f, 13.0f / 6.0f)));
constexpr float LJMAX = (4.0f * 24.0f * EPSILON / SIGMA * (0.216344308307f - 2.0f * 0.0582465445441f));

__device__ float compute_acceleration(float r_angstrom) {
        // in A / s^2
        float temp = powf(SIGMA / r_angstrom, 6); // DON't USE POWF - do the multiplies explicitly
        float acceleration = 24 * EPSILON * (2 * temp * temp - temp) / (r_angstrom * ARGON_MASS);
        //float force = 4 * EPSILON * (12 * pow(SIGMA, 12.0f) / pow(r, 13.0f) - 6 * pow(SIGMA, 6.0f) / pow(r, 7.0f)) / ARGON_MASS;

        return (acceleration < LJMAX) * LJMAX + !(acceleration < LJMAX) * acceleration;
}

__global__ void timestep(float *particle_id, float *src_x, float *src_y, float *src_z,
                         float *vx, float *vy, float *vz, float *dst_x, float *dst_y,
                         float *dst_z, int particle_count)
{
    // each thread gets a particle as a reference particle
    int reference_particle_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (reference_particle_idx >= particle_count)
        return; 

    float reference_x = src_x[reference_particle_idx]; 
    float reference_y = src_y[reference_particle_idx]; 
    float reference_z = src_z[reference_particle_idx]; 

    float ax = 0;
    float ay = 0;
    float az = 0;

    // accumulate accelerations for every other particle
    for (int i = 1; i < particle_count; ++i) {
        float neighbor_x = src_x[(reference_particle_idx + i) % particle_count]; 
        float neighbor_y = src_y[(reference_particle_idx + i) % particle_count]; 
        float neighbor_z = src_z[(reference_particle_idx + i) % particle_count]; 

        float diff_x = reference_x - neighbor_x;
        float diff_y = reference_y - neighbor_y;
        float diff_z = reference_z - neighbor_z;
        float norm = sqrtf((diff_x * diff_x) + (diff_y * diff_y) + (diff_z * diff_z));
        
        float acceleration = compute_acceleration(norm);
        ax += acceleration * (reference_x - neighbor_x) / norm;
        ay += acceleration * (reference_y - neighbor_y) / norm;
        az += acceleration * (reference_z - neighbor_z) / norm;
    }

    // calculate velocity for reference particle
    float reference_vx = vx[reference_particle_idx]; 
    float reference_vy = vy[reference_particle_idx]; 
    float reference_vz = vz[reference_particle_idx]; 
    reference_vx += ax * TIMESTEP_DURATION_FS;
    reference_vy += ay * TIMESTEP_DURATION_FS;
    reference_vz += az * TIMESTEP_DURATION_FS;

    // get new reference particle position taking into account periodic boundary conditions
    float x = reference_x + reference_vx * TIMESTEP_DURATION_FS;
    x += ((x < 0) - (x > UNIVERSE_LENGTH)) * UNIVERSE_LENGTH;
    reference_x = x;
 
    float y = reference_y + reference_vy * TIMESTEP_DURATION_FS;
    y += ((y < 0) - (y > UNIVERSE_LENGTH)) * UNIVERSE_LENGTH;
    reference_y = y;

    float z = reference_z + reference_vz * TIMESTEP_DURATION_FS;
    z += ((z < 0) - (z > UNIVERSE_LENGTH)) * UNIVERSE_LENGTH;
    reference_z = z;

    vx[reference_particle_idx] = reference_vx;
    vy[reference_particle_idx] = reference_vy;
    vz[reference_particle_idx] = reference_vz;
    dst_x[reference_particle_idx] = reference_x;
    dst_y[reference_particle_idx] = reference_y;
    dst_z[reference_particle_idx] = reference_z;
}

int main(int argc, char **argv) 
{
    if (argc != 3) {
        printf("Usage: ./nsquared <input_file> <output_file>\n");
        return 1; 
    }
    
    char *input_file = argv[1];
    char *output_file = argv[2];

    int particle_count;

    float *host_particle_ids = NULL;
    float *host_x = NULL;
    float *host_y = NULL;
    float *host_z = NULL;

    float *device_particle_ids;
    float *device_x_1;
    float *device_y_1;
    float *device_z_1;
    float *device_x_2;
    float *device_y_2;
    float *device_z_2;
    float *vx;
    float *vy;
    float *vz;

    import_atoms(input_file, host_particle_ids, host_x, host_y, host_z, &particle_count);

    GPU_PERROR(hipMalloc(&device_particle_ids, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_x_1, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_y_1, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_z_1, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_x_2, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_y_2, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&device_z_2, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&vx, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&vy, particle_count * sizeof(float)));
    GPU_PERROR(hipMalloc(&vz, particle_count * sizeof(float)));

    GPU_PERROR(hipMemcpy(&device_particle_ids, host_particle_ids, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemcpy(device_x_1, host_x, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemcpy(device_y_1, host_x, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemcpy(device_z_1, host_x, particle_count * sizeof(float), hipMemcpyHostToDevice));
    GPU_PERROR(hipMemset(vx, 0, particle_count * sizeof(float)));
    GPU_PERROR(hipMemset(vy, 0, particle_count * sizeof(float)));
    GPU_PERROR(hipMemset(vz, 0, particle_count * sizeof(float)));

    // set parameters
    dim3 numBlocks((particle_count - 1) / MAX_PARTICLES_PER_BLOCK + 1);
    dim3 threadsPerBlock(MAX_PARTICLES_PER_BLOCK);

#ifdef SIMULATE
    FILE *out = fopen(output_file, "w");
    fprintf(out, "particle_id,x,y,z\n");
#endif

#ifdef TIME_RUN
    struct timespec time_start;
    struct timespec time_stop;
    clock_gettime(CLOCK_REALTIME, &time_start);
#endif

    for (int t = 1l; t <= TIMESTEPS; ++t) {
        if (t % 2 == 1) {
            timestep<<<numBlocks, threadsPerBlock>>>(device_particle_ids, device_x_1, device_y_1, device_z_1, vx, vy, vz, device_x_2, device_y_2, device_z_2, particle_count);
#ifdef SIMULATE
            GPU_PERROR(hipMemcpy(host_x, device_x_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_y, device_y_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_z, device_z_2, particle_count * sizeof(float), hipMemcpyDeviceToHost));
#endif
        } else {
            timestep<<<numBlocks, threadsPerBlock>>>(device_particle_ids, device_x_2, device_y_2, device_z_2, vx, vy, vz, device_x_1, device_y_1, device_z_1, particle_count);
#ifdef SIMULATE
            GPU_PERROR(hipMemcpy(host_x, device_x_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_y, device_y_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
            GPU_PERROR(hipMemcpy(host_z, device_z_1, particle_count * sizeof(float), hipMemcpyDeviceToHost));
#endif
        }
#ifdef SIMULATE
        for (int i = 0; i < particle_count; ++i) {
            fprintf(out, "%d,%f,%f,%f\n", host_particle_ids[i], host_x[i], host_y[i], host_z[i]);
        }
        fprintf(out, "\n");
#endif
    }

#ifdef TIME_RUN
    clock_gettime(CLOCK_REALTIME, &time_stop);

    struct timespec temp;
    temp.tv_sec = time_stop.tv_sec - time_start.tv_sec;
    temp.tv_nsec = time_stop.tv_nsec - time_start.tv_nsec;
    if (temp.tv_nsec < 0) {
        temp.tv_sec = temp.tv_sec - 1;
        temp.tv_nsec = temp.tv_nsec + 1000000000;
    }

    printf("nsquared,%f\n", ((double) temp.tv_sec) + (((double) temp.tv_nsec) * 1e-9));

    if (TIMESTEPS & 1) {
        GPU_PERROR(hipMemcpy(host_x, device_x_2, particle_count * sizeof(struct Particle), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_y, device_y_2, particle_count * sizeof(struct Particle), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_z, device_z_2, particle_count * sizeof(struct Particle), hipMemcpyDeviceToHost));
    } else {
        GPU_PERROR(hipMemcpy(host_x, device_x_1, particle_count * sizeof(struct Particle), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_y, device_y_1, particle_count * sizeof(struct Particle), hipMemcpyDeviceToHost));
        GPU_PERROR(hipMemcpy(host_z, device_z_1, particle_count * sizeof(struct Particle), hipMemcpyDeviceToHost));
    }
        
    FILE *out = fopen(output_file, "w");
    fprintf(out, "particle_id,x,y,z\n");
    for (int i = 0; i < particle_count; ++i) {
        fprintf(out, "%d,%f,%f,%f\n", out_list[i].particle_id, out_list[i].x, out_list[i].y, out_list[i].z);
    }
#endif

    GPU_PERROR(hipFree(device_particle_ids));
    GPU_PERROR(hipFree(device_x_1));
    GPU_PERROR(hipFree(device_y_1));
    GPU_PERROR(hipFree(device_z_1));
    GPU_PERROR(hipFree(device_x_2));
    GPU_PERROR(hipFree(device_y_2));
    GPU_PERROR(hipFree(device_z_2));
    GPU_PERROR(hipFree(vx));
    GPU_PERROR(hipFree(vy));
    GPU_PERROR(hipFree(vz));

    return 0;
}
