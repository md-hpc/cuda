
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand.h>


// minimum max shared memory size per SM across all architectures is 64K
// minimum max resident block per SM across all architectures is 16
// so worst case, each block will have max 4K shared memory

// use profiler to identify optimal size ie. CUDA occupancy API, nvvp
#define NUM_PARTICLES 3
#define MAX_PARTICLES_PER_CELL 128

#define CELL_CUTOFF_RADIUS_ANGST 100
#define CELL_LENGTH_X 3
#define CELL_LENGTH_Y 3
#define CELL_LENGTH_Z 3

#define TIMESTEPS 1
#define TIMESTEP_DURATION_FS 1                            
#define EPSILON 1.0f
#define SIGMA 1.0f
#define LJMIN (-4.0f * 24.0f * EPSILON / SIGMA * (powf(7.0f / 26.0f, 7.0f / 6.0f) - 2.0f * powf(7.0f / 26.0f, 13.0f / 6.0f)))

#define PLUS_1(dimension, length) ((dimension != length - 1) * (dimension + 1))
#define MINUS_1(dimension, length) ((dimension == 0) * length + dimension - 1)
#define GPU_PERROR(err) do {\
    if (err != hipSuccess) {\
        fprintf(stderr,"gpu_perror: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);\
        exit(err);\
    }\
} while (0);
	


// particle stores coordinates and velocities in x,y,z dimensions
struct Particle {
    int particle_id;
    float x;
    float y;
    float z;
    float vx;
    float vy;
    float vz;
};

// cell is an array of particles
struct Cell {
    struct Particle particle_list[MAX_PARTICLES_PER_CELL];
};

// LJ force computation
__device__ float compute_acceleration(float r1, float r2) {
    float r = fabsf(r1 - r2);
    float force = 4 * EPSILON * (6 * powf(SIGMA, 6.0f) / powf(r, 7.0f) - 12 * powf(SIGMA, 12.0f) / powf(r, 13.0f));

    return (force < LJMIN) * LJMIN + !(force < LJMIN) * force;
}

// the meat:
__global__ void force_eval(struct Cell *cell_list, float *accelerations)
{
    /*
        1D block array will look like this:
                14               14
        | HNNNNNNNNNNNNN | HNNNNNNNNNNNNN | ... | 
            0               14               28    X*Y*Z*14

        Map one block to a home-neighbor tuple (home cell, neighbor cell)
        Map one thread to a particle index in the home cell, which calculates accelerations in 
        a one to all fashion with the particles in the neighbor cell.

        CAREFUL: one of the home-neighbor tuple will actually be a home-home tuple
    */

    // find hcell coordinate based off of block index
    int home_x = blockIdx.x % CELL_LENGTH_X;
    int home_y = blockIdx.x / CELL_LENGTH_X % CELL_LENGTH_Y;
    int home_z = blockIdx.x / (CELL_LENGTH_Y * CELL_LENGTH_X) % CELL_LENGTH_Z;

    // trust me on this :)
    // turns out, we don't need branchless programming here because all threads in a warp and all warps in a block will follow the same branch
    // TODO: remove branchless programming here
    int neighbor_x = (blockIdx.y < 9) * PLUS_1(home_x, CELL_LENGTH_X)
                   + (blockIdx.y >= 9) * home_x;
    int neighbor_y = (blockIdx.y < 3) * MINUS_1(home_y, CELL_LENGTH_Y)
                   + (blockIdx.y >= 3 && blockIdx.y <= 5 || blockIdx.y > 11) * home_y
                   + (blockIdx.y >= 6 && blockIdx.y <= 11) * PLUS_1(home_y, CELL_LENGTH_Y);
    int neighbor_z = (blockIdx.y % 3 == 0) * PLUS_1(home_z, CELL_LENGTH_Z)
                   + (blockIdx.y % 3 == 1) * home_z
                   + (blockIdx.y % 3 == 2) * MINUS_1(home_z, CELL_LENGTH_Z);

    // define and assign shared memory
    __shared__ struct Cell neighbor_cell;
    neighbor_cell.particle_list[threadIdx.x] = cell_list[neighbor_x + neighbor_y * CELL_LENGTH_X + neighbor_z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[threadIdx.x];

    // set the particle thread is assigned to from particle from hcell
    // TODO: can optimize by splitting struct into positions and velocities and only memcpy the positions
    struct Particle reference_particle = cell_list[home_x + home_y * CELL_LENGTH_X + home_z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[threadIdx.x];

    // synchronizes threads within a block (all threads must complete tasks)
    __syncthreads();

    // TODO: probably can optimize using shared acceleration for home cell particles
    // TODO: probably can optimize using warp indexing
    // if particle exists loop through every particle in ncell particle list
    if (reference_particle.particle_id != -1) {
        //TODO: (easy) write the rest of the force computation
        int neighbor_is_home = home_x == neighbor_x && home_y == neighbor_y && home_z == neighbor_z;
        for (int i = 0; i < MAX_PARTICLES_PER_CELL; ++i) {
            if (neighbor_cell.particle_list[i].particle_id == -1)
                break;

            if (neighbor_is_home && !(reference_particle.x < neighbor_cell.particle_list[i].x))
                continue;

            int neighbor_particle_id = neighbor_cell.particle_list[i].particle_id;

            float ax, ay, az;
            if (neighbor_is_home) {
                ax = compute_acceleration(reference_particle.x, neighbor_cell.particle_list[i].x);
                ay = compute_acceleration(reference_particle.y, neighbor_cell.particle_list[i].y);
                az = compute_acceleration(reference_particle.z, neighbor_cell.particle_list[i].z);
            } else {
                // boolean expression can be optimized knowing the fact that one dimension of the neighboring half shell is only +1 and not -1
                // for periodic boundary condition
                float neighbor_particle_virtual_x = neighbor_cell.particle_list[i].x + ((home_x - neighbor_x == CELL_LENGTH_X - 1) + (neighbor_x - home_x == CELL_LENGTH_X - 1) * -1) * (CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST);
                float neighbor_particle_virtual_y = neighbor_cell.particle_list[i].y + ((home_y - neighbor_y == CELL_LENGTH_Y - 1) + (neighbor_y - home_y == CELL_LENGTH_Y - 1) * -1) * (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST);
                float neighbor_particle_virtual_z = neighbor_cell.particle_list[i].z + ((home_z - neighbor_z == CELL_LENGTH_Z - 1) + (neighbor_z - home_z == CELL_LENGTH_Z - 1) * -1) * (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST);

                // computed accelerations
                ax = compute_acceleration(reference_particle.x, neighbor_particle_virtual_x);
                ay = compute_acceleration(reference_particle.y, neighbor_particle_virtual_y);
                az = compute_acceleration(reference_particle.z, neighbor_particle_virtual_z);
            }

            atomicAdd(&accelerations[reference_particle.particle_id * 3], ax);
            atomicAdd(&accelerations[reference_particle.particle_id * 3 + 1], ay);
            atomicAdd(&accelerations[reference_particle.particle_id * 3 + 2], az);

            atomicAdd(&accelerations[neighbor_particle_id * 3], -ax);
            atomicAdd(&accelerations[neighbor_particle_id * 3 + 1], -ay);
            atomicAdd(&accelerations[neighbor_particle_id * 3 + 2], -az);
        }
    }
}

__global__ void particle_update(struct Cell *cell_list, float *accelerations)
{
    // 1 block -> 1 cell
    // 1 thread -> 1 particle

    int reference_particle_id = cell_list[blockIdx.x].particle_list[threadIdx.x].particle_id;
    if (reference_particle_id == -1)
        return;

    cell_list[blockIdx.x].particle_list[threadIdx.x].vx += accelerations[reference_particle_id * 3] * TIMESTEP_DURATION_FS;
    cell_list[blockIdx.x].particle_list[threadIdx.x].vy += accelerations[reference_particle_id * 3 + 1] * TIMESTEP_DURATION_FS;
    cell_list[blockIdx.x].particle_list[threadIdx.x].vz += accelerations[reference_particle_id * 3 + 2] * TIMESTEP_DURATION_FS;

    float x = cell_list[blockIdx.x].particle_list[threadIdx.x].x + cell_list[blockIdx.x].particle_list[threadIdx.x].vx * TIMESTEP_DURATION_FS;
    x += (x < 0) * (CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST) + (x > CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST) * -(CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST);
    cell_list[blockIdx.x].particle_list[threadIdx.x].x = x;

    float y = cell_list[blockIdx.x].particle_list[threadIdx.x].y + cell_list[blockIdx.x].particle_list[threadIdx.x].vy * TIMESTEP_DURATION_FS;
    y += (y < 0) * (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST) + (y > CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST) * -(CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST);
    cell_list[blockIdx.x].particle_list[threadIdx.x].y = y;

    float z = cell_list[blockIdx.x].particle_list[threadIdx.x].z + cell_list[blockIdx.x].particle_list[threadIdx.x].vz * TIMESTEP_DURATION_FS;
    z += (z < 0) * (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST) + (z > CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST) * -(CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST);
    cell_list[blockIdx.x].particle_list[threadIdx.x].z = z;

    accelerations[reference_particle_id] = 0;
}

// update cell lists because particles have moved
__global__ void motion_update(struct Cell *cell_list_src, struct Cell *cell_list_dst)
{
    /*
        1 block per cell
        right now 1 thread per block
        1 thread per particle list
        keeps counter on next free spot on new particle list
        once a -1 in the old particle list is reached, there are no particles to the right
    */
    // get home cell coordinates

    // threadIdx.x is always 0 because we are indexing by blockIdx.x
    int home_x = blockIdx.x % CELL_LENGTH_X;
    int home_y = blockIdx.x / CELL_LENGTH_X % CELL_LENGTH_Y;
    int home_z = blockIdx.x / (CELL_LENGTH_X * CELL_LENGTH_Y) % CELL_LENGTH_Z;

    // location of where thread is in buffer
    int free_idx = 0;

    for (int current_cell_idx = 0; current_cell_idx < CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z; ++current_cell_idx) {
        for (int particle_idx = 0; particle_idx < MAX_PARTICLES_PER_CELL && cell_list_src[current_cell_idx].particle_list[particle_idx].particle_id != -1; ++particle_idx) {
            struct Particle current_particle = cell_list_src[current_cell_idx].particle_list[particle_idx];
            int new_cell_x = current_particle.x / (CELL_LENGTH_X * CELL_CUTOFF_RADIUS_ANGST);
            int new_cell_y = current_particle.y / (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS_ANGST);
            int new_cell_z = current_particle.z / (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS_ANGST);

            if (home_x == new_cell_x && home_y == new_cell_y && home_z == new_cell_z) {
                cell_list_dst[home_x + home_y * CELL_LENGTH_X + home_z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[free_idx++] = current_particle;
            }
        }
    }
    cell_list_dst[home_x + home_y * CELL_LENGTH_X + home_z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[free_idx].particle_id = -1;
}

// initialize cells with random particle data
void initialize_cell_list(struct Cell cellList[CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z])
{
    // initialize cell list, -1 for empty cell
    memset(cellList, -1, sizeof(struct Cell)*CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z);
    for (int i = 0; i < NUM_PARTICLES; ++i) {
        int x = rand() % CELL_LENGTH_X;
        int y = rand() % CELL_LENGTH_Y;
        int z = rand() % CELL_LENGTH_Z;
        // assign random particle data
        struct Particle particle = {
            .particle_id = i,
            .x = x * CELL_CUTOFF_RADIUS_ANGST + ((float) rand() / RAND_MAX) * CELL_CUTOFF_RADIUS_ANGST,
            .y = y * CELL_CUTOFF_RADIUS_ANGST + ((float) rand() / RAND_MAX) * CELL_CUTOFF_RADIUS_ANGST,
            .z = z * CELL_CUTOFF_RADIUS_ANGST + ((float) rand() / RAND_MAX) * CELL_CUTOFF_RADIUS_ANGST,
            .vx = 0,
            .vy = 0,
            .vz = 0,
        };
        // copy particle to to cell list
        for (int j = 0; j < MAX_PARTICLES_PER_CELL; ++j) {
            if (cellList[x + y * CELL_LENGTH_X + z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[j].particle_id == -1) {
                memcpy(&cellList[x + y * CELL_LENGTH_X + z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[j], &particle, sizeof(struct Particle));
                break;
            }
        }
    }
    for (int x = 0; x < CELL_LENGTH_X; ++x) {
        for (int y = 0; y < CELL_LENGTH_Y; ++y) {
            for (int z = 0; z < CELL_LENGTH_Z; ++z) {
                int count = 0;
                while (cellList[x + y * CELL_LENGTH_X + z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[count].particle_id != -1) {
                    printf("%d: (%f, %f, %f)\n", cellList[x + y * CELL_LENGTH_X + z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[count].particle_id, cellList[x + y * CELL_LENGTH_X + z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[count].x , cellList[x + y * CELL_LENGTH_X + z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[count].y, cellList[x + y * CELL_LENGTH_X + z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[count].z);
                    count++;
                }
            }
        }
    }
}

int main() 
{
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // INITIALIZE CELL LIST WITH PARTICLE DATA
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // initialize (or import) particle data for simulation
    struct Cell cell_list[CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z];
    memset(cell_list, -1, sizeof(struct Cell)*CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z);
    //initialize_cell_list(cell_list);
    cell_list[0].particle_list[0].particle_id = 0;
    cell_list[0].particle_list[0].x = 0;
    cell_list[0].particle_list[0].y = 0;
    cell_list[0].particle_list[0].z = 0;
    cell_list[0].particle_list[0].vx = 0;
    cell_list[0].particle_list[0].vy = 0;
    cell_list[0].particle_list[0].vz = 0;

    cell_list[0].particle_list[1].particle_id = 1;
    cell_list[0].particle_list[1].x = 1;
    cell_list[0].particle_list[1].y = 1;
    cell_list[0].particle_list[1].z = 1;
    cell_list[0].particle_list[1].vx = 0;
    cell_list[0].particle_list[1].vy = 0;
    cell_list[0].particle_list[1].vz = 0;
    // device_cell_list stores an array of Cells, where each Cell contains a particle_list
    struct Cell *device_cell_list1;
    struct Cell *device_cell_list2;
    // cudaMalloc initializes GPU global memory to be used as parameter for GPU kernel
    GPU_PERROR(hipMalloc(&device_cell_list1, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell)));
    GPU_PERROR(hipMalloc(&device_cell_list2, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell)));
    GPU_PERROR(hipMemcpy(device_cell_list1, cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyHostToDevice));


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // INITIALIZE ACCELERATIONS
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /*
        accelerations stores accelerations (in x y z dimensions) of each particle to be used in motion update.
        - index of accelerations is related to particle_id
        - particle_id * 3 gives index of accelerations for x dimension
        - (particle_id * 3) + 1 gives index of y
        - (particle_id * 3) + 2 gives index of y
    */
    float *accelerations;
    GPU_PERROR(hipMalloc(&accelerations, NUM_PARTICLES * 3 * sizeof(float)));
    GPU_PERROR(hipMemset(accelerations, 0, NUM_PARTICLES * 3 * sizeof(float)));


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // INITIALIZE PARAMETERS FOR FORCE COMPUTATION AND MOTION UPDATE
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // defines block and thread dimensions
    // dim3 is an integer vector type most commonly used to pass the grid and block dimensions in a kernel invocation [X x Y x Z]
    // there are 2^31 blocks in x dimension while y and z have at most 65536 blocks
    dim3 numBlocksForce(CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z, 14);        // (CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * 14) x 1 x 1
    dim3 numBlocksParticle(CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z);
    dim3 numBlocksMotion(CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z);            // (CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z) x 1 x 1
    dim3 threadsPerBlockForce(MAX_PARTICLES_PER_CELL);                              // MAX_PARTICLES_PER_CELL x 1 x 1
    dim3 threadsPerBlockParticle(MAX_PARTICLES_PER_CELL);                              // MAX_PARTICLES_PER_CELL x 1 x 1
//    dim3 threadsPerBlockMotion(CELL_LENGTH_X, CELL_LENGTH_Y, CELL_LENGTH_Z);  

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // FORCE COMPUTATION AND MOTION UPDATE
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // do force evaluation and motion update for each time step
    // steps are separated to ensure threads are synchronized (that force_eval is done)
    // output of force_eval is stores in device_cell_list and accelerations

    int flag = 1;

    // address + ((flag == 1) * sizeof(cell list))

    // if flag == 0, then pass in address
    // if flag == 1, then pass in address + offset
    // flag = !flag;

    for (int t = 0; t < TIMESTEPS; ++t) {
        if (flag) {
            force_eval<<<numBlocksForce, threadsPerBlockForce>>>(device_cell_list1, accelerations);
            particle_update<<<numBlocksParticle, threadsPerBlockParticle>>>(device_cell_list1, accelerations);
            motion_update<<<numBlocksMotion, 1>>>(device_cell_list1, device_cell_list2);
        } else {
            force_eval<<<numBlocksForce, threadsPerBlockForce>>>(device_cell_list2, accelerations);
            particle_update<<<numBlocksParticle, threadsPerBlockParticle>>>(device_cell_list2, accelerations);
            motion_update<<<numBlocksMotion, 1>>>(device_cell_list2, device_cell_list1);
        }
        flag = !flag;
    }

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //  COPY FINAL RESULT BACK TO HOST CPU
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // if flag == 0, then results are in the second half
    // if flag == 1, then results are in the first half
    if (flag) {
        GPU_PERROR(hipMemcpy(cell_list, device_cell_list1, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyDeviceToHost));
    } else {
        GPU_PERROR(hipMemcpy(cell_list, device_cell_list2, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyDeviceToHost));
    }
    GPU_PERROR(hipFree(device_cell_list1));
    GPU_PERROR(hipFree(device_cell_list2));

    for (int x = 0; x < CELL_LENGTH_X; ++x) {
        for (int y = 0; y < CELL_LENGTH_Y; ++y) {
            for (int z = 0; z < CELL_LENGTH_Z; ++z) {
                int count = 0;
                while (cell_list[x + y * CELL_LENGTH_X + z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[count].particle_id != -1) {
                    printf("%d: (%f, %f, %f)\n", cell_list[x + y * CELL_LENGTH_X + z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[count].particle_id, cell_list[x + y * CELL_LENGTH_X + z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[count].x , cell_list[x + y * CELL_LENGTH_X + z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[count].y, cell_list[x + y * CELL_LENGTH_X + z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[count].z);
                    count++;
                }
            }
        }
    }

    return 0;
}
