#include "hip/hip_runtime.h"
#include <stdio.h>

// use profiler to identify optimal size ie. CUDA occupancy API, nvvp
#define NUM_PARTICLES 5
#define MAX_PARTICLES_PER_CELL 128

#define CELL_CUTOFF_RADIUS 1f
#define CELL_LENGTH_X 3
#define CELL_LENGTH_Y 3
#define CELL_LENGTH_Z 3

#define TIMESTEPS 1
#define TIMESTEP_DURATION 1                            
#define EPSILON 1
#define SIGMA 1

#define PLUS_1(dimension, length) ((dimension != length - 1) * (dimension + 1))
#define MINUS_1(dimension, length) ((dimension == 0) * length + dimension - 1)

// particle stores coordinates and velocities in x,y,z dimensions
struct Particle {
    int particle_id;
    float x;
    float y;
    float z;
    float vx;
    float vy;
    float vz;
};

// cell is an array of particles
struct Cell {
    struct Particle particle_list[MAX_PARTICLES_PER_CELL];
};

// force computation
__device__ float compute_force(float x1, float x2) {
    float force = 10;
    return force;
}
// the meat:
__global__ void force_eval(struct Cell *cell_list, float *accelerations)
{
    /*
        1D block array will look like this:
                14               14
        | HNNNNNNNNNNNNN | HNNNNNNNNNNNNN | ... | 
         0               14               28    X*Y*Z*14

        Map one block to a home-neighbor tuple (home cell, neighbor cell)
        Map one thread to a particle index in the home cell, which calculates accelerations in 
        a one to all fashion with the particles in the neighbor cell.

        CAREFUL: one of the home-neighbor tuple will actually be a home-home tuple
    */

    int home_x = blockIdx.x / (14 * CELL_LENGTH_Y * CELL_LENGTH_Z) % CELL_LENGTH_X;
    int home_y = blockIdx.x / (14 * CELL_LENGTH_Z) % CELL_LENGTH_Y;
    int home_z = blockIdx.x / 14 % CELL_LENGTH_Z;

    // trust me on this :)
    int local_idx = blockIdx.x % 14;
    int neighbor_x = (local_idx < 9) * PLUS_1(home_x, CELL_LENGTH_X)
                   + (local_idx >= 9) * home_x;
    int neighbor_y = (local_idx < 3) * MINUS_1(home_y, CELL_LENGTH_Y)
                   + (local_idx >= 3 && local_idx <= 5 || local_idx > 11) * home_y
                   + (local_idx >= 6 && local_idx <= 11) * PLUS_1(home_y, CELL_LENGTH_Y);
    int neighbor_z = (local_idx % 3 == 0) * PLUS_1(home_z, CELL_LENGTH_Z)
                   + (local_idx % 3 == 1) * home_z
                   + (local_idx % 3 == 2) * MINUS_1(home_z, CELL_LENGTH_Z);

    // define and assign shared memory
    __shared__ struct Cell home_cell;
    __shared__ struct Cell neighbor_cell;
    home_cell.particle_list[threadIdx.x] = cell_list[home_x][home_y][home_z].particle_list[threadIdx.x];
    neighbor_cell.particle_list[threadIdx.x] = cell_list[neighbor_x][neighbor_y][neighbor_z].particle_list[threadIdx.x];

    int reference_particle_id = home_cell.particle_list[threadIdx.x].particle_id;
    if (reference_particle_id == -1)
        return;

    __syncthreads();

    //TODO: (easy) write the rest of the force computation
    for (int i = 0; neighbor_cell.particle_list[i].particle_id != -1 && i < MAX_PARTICLES_PER_CELL; ++i) {
        // boolean expression can be optimized knowing the fact that one dimension of the neighboring half shell is only +1 and not -1
        float neighbor_particle_virtual_x = neighbor_cell.particle_list[i].x + ((home_x - neighbor_x == CELL_LENGTH_X - 1) + (neighbor_x - home_x == CELL_LENGTH_X - 1) * -1) * (CELL_LENGTH_X * CELL_CUTOFF_RADIUS);
        float neighbor_particle_virtual_y = neighbor_cell.particle_list[i].y + ((home_y - neighbor_y == CELL_LENGTH_Y - 1) + (neighbor_y - home_y == CELL_LENGTH_Y - 1) * -1) * (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS);
        float neighbor_particle_virtual_z = neighbor_cell.particle_list[i].z + ((home_z - neighbor_z == CELL_LENGTH_Z - 1) + (neighbor_z - home_z == CELL_LENGTH_Z - 1) * -1) * (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS);

        // can probably optimize using linear algebras
        atomicAdd(&accelerations[reference_particle_id][0], compute_force(home_cell.particle_list[threadIdx.x].x, neighbor_particle_virtual_x));
        atomicAdd(&accelerations[reference_particle_id][1], compute_force(home_cell.particle_list[threadIdx.x].y, neighbor_particle_virtual_y));
        atomicAdd(&accelerations[reference_particle_id][2], compute_force(home_cell.particle_list[threadIdx.x].z, neighbor_particle_virtual_z));
    }

    // choose one block to "work" on the home cell
    // threads update their associated particle here
    // all particles are still in their original cell
    if (local_idx != 0)
        return;
    home_cell.particle_list[threadIdx.x].vx += accelerations[reference_particle_id] * TIMESTEP_DURATION;
    home_cell.particle_list[threadIdx.x].vy += accelerations[reference_particle_id + 1] * TIMESTEP_DURATION;
    home_cell.particle_list[threadIdx.x].vz += accelerations[reference_particle_id + 2] * TIMESTEP_DURATION;
    home_cell.particle_list[threadIdx.x].x = (home_cell.particle_list[threadIdx.x].x + home_cell.particle_list[threadIdx.x].vx * TIMESTEP_DURATION) - (CELL_LENGTH_X * CELL_CUTOFF_RADIUS) * floor(home_cell.particle_list[threadIdx.x].x / (CELL_LENGTH_X * CELL_CUTOFF_RADIUS));
    home_cell.particle_list[threadIdx.x].y = (home_cell.particle_list[threadIdx.x].y + home_cell.particle_list[threadIdx.x].vy * TIMESTEP_DURATION) - (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS) * floor(home_cell.particle_list[threadIdx.x].y / (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS));
    home_cell.particle_list[threadIdx.x].z = (home_cell.particle_list[threadIdx.x].z + home_cell.particle_list[threadIdx.x].vz * TIMESTEP_DURATION) - (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS) * floor(home_cell.particle_list[threadIdx.x].z / (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS));

    // update global
    cell_list[home_z + home_y * CELL_LENGTH_Z + home_x * CELL_LENGTH_Z * CELL_LENGTH_Y].particle_list[threadIdx.x] = cell.particle_list[threadIdx.x];
}

// update cell lists because particles have moved
__global__ void motion_update(struct Cell *cell_list, float *accelerations)
{
    // get home cell coordinates
    int home_x = blockIdx.x % CELL_LENGTH_X;
    int home_y = blockIdx.x / CELL_LENGTH_X % CELL_LENGTH_Y;
    int home_z = blockIdx.x / (CELL_LENGTH_X * CELL_LENGTH_Y) % CELL_LENGTH_Z;

    __shared__ struct Cell cell;
    cell.particle_list[threadIdx.x] = cell_list[cell_x + cell_y * CELL_LENGTH_X + cell_z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[threadIdx.x];

    int particle_id = cell.particle_list[threadIdx.x].particle_id;

    // update cell list with updated particles
    // one block per cell
    // one thread per cell not equal to block's cell
    // that thread loops over each particle and copies it to home cell's particle list
    int new_cell_x = cell_particle_list[threadIdx.x].x / (CELL_LENGTH_X * CELL_CUTOFF_RADIUS);
    int new_cell_y = cell_particle_list[threadIdx.x].y / (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS);
    int new_cell_z = cell_particle_list[threadIdx.x].z / (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS);

    accelerations[particle_id] = 0;
    accelerations[particle_id + 1] = 0;
    accelerations[particle_id + 2] = 0;
}

// initialize cells with random particle data
void initialize_cell_list(struct Cell cellList[CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z])
{
        // initialize cell list, -1 for empty cell
        memset(cellList, -1, sizeof(cellList));
        for (int i = 0; i < NUM_PARTICLES; ++i) {
                int x = rand() % CELL_LENGTH_X;
                int y = rand() % CELL_LENGTH_Y;
                int z = rand() % CELL_LENGTH_Z;

                // assign random particle data
                struct Particle particle = {
                        .particle_id = i,
                        .x = x * CELL_CUTOFF_RADIUS + ((float) rand() / RAND_MAX) * CELL_CUTOFF_RADIUS,
                        .y = y * CELL_CUTOFF_RADIUS + ((float) rand() / RAND_MAX) * CELL_CUTOFF_RADIUS,
                        .z = z * CELL_CUTOFF_RADIUS + ((float) rand() / RAND_MAX) * CELL_CUTOFF_RADIUS,
                        .vx = 0,
                        .vy = 0,
                        .vz = 0,
                };
                // copy particle to to cell list
                for (int j = 0; j < MAX_PARTICLES_PER_CELL; ++j) {
                    if (cellList[x][y][z].particle_list[j].particle_id == -1) {
                        memcpy(&cellList[x][y][z].particle_list[j], &particle, sizeof(struct Particle));
                        break;
                    }
                }
        }
}

int main() 
{
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // INITIALIZE CELL LIST WITH PARTICLE DATA
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // initialize (or import) particle data for simulation
    struct Cell cell_list[CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z];
    initialize_cell_list(&cell_list);
    // device_cell_list stores an array of Cells, where each Cell contains a particle_list
    struct Cell *device_cell_list;
    // hipMalloc initializes GPU global memory to be used as parameter for GPU kernel
    hipMalloc(&device_cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell));
    hipMemcpy(device_cell_list, cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyHostToDevice);


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // INITIALIZE ACCELERATIONS
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /*
        accelerations stores accelerations (in x y z dimensions) of each particle to be used in motion update.
        - index of accelerations is related to particle_id
        - particle_id * 3 gives index of accelerations for x dimension
        - (particle_id * 3) + 1 gives index of y
        - (particle_id * 3) + 2 gives index of y
    */
    float *accelerations;
    hipMalloc(&accelerations, MAX_PARTICLES_PER_CELL * 3 * sizeof(float));
    hipMemset(accelerations, 0, MAX_PARTICLES_PER_CELL * 3 * sizeof(float));


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // INITIALIZE PARAMETERS FOR FORCE COMPUTATION AND MOTION UPDATE
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // defines block and thread dimensions
    // dim3 is an integer vector type most commonly used to pass the grid and block dimensions in a kernel invocation [X x Y x Z]
    // there are 2^31 blocks in x dimension while y and z have at most 65536 blocks
    dim3 numBlocksForce(CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * 14);        // (CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * 14) x 1 x 1
    dim3 numBlocksMotion(CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z);            // (CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z) x 1 x 1
    dim3 threadsPerBlockForce(MAX_PARTICLES_PER_CELL);                              // MAX_PARTICLES_PER_CELL x 1 x 1
    dim3 threadsPerBlockMotion(CELL_LENGTH_X, CELL_LENGTH_Y, CELL_LENGTH_Z);  

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // FORCE COMPUTATION AND MOTION UPDATE
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // do force evaluation and motion update for each time step
    // steps are separated to ensure threads are synchronized (that force_eval is done)
    // output of force_eval is stores in device_cell_list and accelerations
    for (int t = 0; t < TIMESTEPS; ++t) {
        force_eval<<<numBlocksForce, threadsPerBlockForce>>>(device_cell_list, accelerations);
        motion_update<<<numBlocksMotion, threadsPerBlockMotion>>>(device_cell_list, accelerations);
    }

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //  COPY FINAL RESULT BACK TO HOST CPU
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    hipMemcpy(cell_list, device_cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * struct(struct Cell), hipMemcpyDeviceToHost);
    hipFree(device_cell_list);

    return 0;
}
