#include "hip/hip_runtime.h"
#include <stdio.h>

// use profiler to identify optimal size ie. CUDA occupancy API, nvvp
#define NUM_PARTICLES 5
#define MAX_PARTICLES_PER_CELL 128

#define CELL_CUTOFF_RADIUS 1f
#define CELL_LENGTH_X 3
#define CELL_LENGTH_Y 3
#define CELL_LENGTH_Z 3

#define TIMESTEPS 1
#define TIMESTEP_DURATION 1                            
#define EPSILON 1
#define SIGMA 1

#define PLUS_1(dimension, length) ((dimension != length - 1) * (dimension + 1))
#define MINUS_1(dimension, length) ((dimension == 0) * length + dimension - 1)

// particle stores coordinates and velocities in x,y,z dimensions
struct Particle {
    int particleId;
    float x;
    float y;
    float z;
    float vx;
    float vy;
    float vz;
};

// cell is an array of particles
struct Cell {
    struct Particle particle_list[MAX_PARTICLES_PER_CELL];
};

// force computation
__device__ float compute_force(float x1, float x2) {
    float force = 10;
    return force;
}
// the meat:
__global__ void force_eval(struct Cell *cell_list, float *accelerations)
{
    /*
        1D block array will look like this:
                14               14
        | HNNNNNNNNNNNNN | HNNNNNNNNNNNNN | ... | 
         0               14               28    X*Y*Z*14

        Map one block to a home-neighbor tuple (home cell, neighbor cell)
        Map one thread to a particle index in the home cell, which calculates accelerations in 
        a one to all fashion with the particles in the neighbor cell.

        CAREFUL: one of the home-neighbor tuple will actually be a home-home tuple
    */

    int home_x = blockIdx.x / (14 * CELL_LENGTH_Y * CELL_LENGTH_Z) % CELL_LENGTH_X;
    int home_y = blockIdx.x / (14 * CELL_LENGTH_Z) % CELL_LENGTH_Y;
    int home_z = blockIdx.x / 14 % CELL_LENGTH_Z;

    // trust me on this :)
    int local_idx = blockIdx.x % 14;
    int neighbor_x = (local_idx < 9) * PLUS_1(home_x, CELL_LENGTH_X)
                   + (local_idx >= 9) * home_x;
    int neighbor_y = (local_idx < 3) * MINUS_1(home_y, CELL_LENGTH_Y)
                   + (local_idx >= 3 && local_idx <= 5 || local_idx > 11) * home_y
                   + (local_idx >= 6 && local_idx <= 11) * PLUS_1(home_y, CELL_LENGTH_Y);
    int neighbor_z = (local_idx % 3 == 0) * PLUS_1(home_z, CELL_LENGTH_Z)
                   + (local_idx % 3 == 1) * home_z
                   + (local_idx % 3 == 2) * MINUS_1(home_z, CELL_LENGTH_Z);

    // define and assign shared memory
    __shared__ struct Cell home_cell;
    __shared__ struct Cell neighbor_cell;
    home_cell.particle_list[threadIdx.x] = cell_list[home_x][home_y][home_z].particle_list[threadIdx.x];
    neighbor_cell.particle_list[threadIdx.x] = cell_list[neighbor_x][neighbor_y][neighbor_z].particle_list[threadIdx.x];

    int reference_particle_id = home_cell.particle_list[threadIdx.x].particleId;
    if (reference_particle_id == -1)
        return;

    __syncthreads();

    //TODO: (easy) write the rest of the force computation
    for (int i = 0; neighbor_cell.particle_list[i].particleId != -1 && i < MAX_PARTICLES_PER_CELL; ++i) {
        // boolean expression can be optimized knowing the fact that one dimension of the neighboring half shell is only +1 and not -1
        float neighbor_particle_virtual_x = neighbor_cell.particle_list[i].x + ((home_x - neighbor_x == CELL_LENGTH_X - 1) + (neighbor_x - home_x == CELL_LENGTH_X - 1) * -1) * (CELL_LENGTH_X * CELL_CUTOFF_RADIUS);
        float neighbor_particle_virtual_y = neighbor_cell.particle_list[i].y + ((home_y - neighbor_y == CELL_LENGTH_Y - 1) + (neighbor_y - home_y == CELL_LENGTH_Y - 1) * -1) * (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS);
        float neighbor_particle_virtual_z = neighbor_cell.particle_list[i].z + ((home_z - neighbor_z == CELL_LENGTH_Z - 1) + (neighbor_z - home_z == CELL_LENGTH_Z - 1) * -1) * (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS);

        // can probably optimize using linear algebras
        atomicAdd(&accelerations[reference_particle_id][0], compute_force(home_cell.particle_list[threadIdx.x].x, neighbor_particle_virtual_x));
        atomicAdd(&accelerations[reference_particle_id][1], compute_force(home_cell.particle_list[threadIdx.x].y, neighbor_particle_virtual_y));
        atomicAdd(&accelerations[reference_particle_id][2], compute_force(home_cell.particle_list[threadIdx.x].z, neighbor_particle_virtual_z));
    }

    // particle update here, all particles are still in their original cell
}

__global__ void motion_update(struct Cell *cell_list, float *accelerations)
{
    // think about particles moving from cell to cell
    int cell_x = blockIdx.x % CELL_LENGTH_X;
    int cell_y = blockIdx.x / CELL_LENGTH_X % CELL_LENGTH_Y;
    int cell_z = blockIdx.x / (CELL_LENGTH_X * CELL_LENGTH_Y) % CELL_LENGTH_Z;

    __shared__ struct Cell cell;
    cell.particle_list[threadIdx.x] = cell_list[cell_x + cell_y * CELL_LENGTH_X + cell_z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[threadIdx.x];

    int particleId = cell.particle_list[threadIdx.x].particleId;

    // python: cell_from_position = lambda r: linear_idx(*[floor(x/CUTOFF)%UNIVERSE_SIZE for x in r])
    cell.particle_list[threadIdx.x].vx += accelerations[particleId] * TIMESTEP_DURATION;
    cell.particle_list[threadIdx.x].vy += accelerations[particleId + 1] * TIMESTEP_DURATION;
    cell.particle_list[threadIdx.x].vz += accelerations[particleId + 2] * TIMESTEP_DURATION;
    cell.particle_list[threadIdx.x].x = (cell.particle_list[threadIdx.x].x + cell.particle_list[threadIdx.x].vx * TIMESTEP_DURATION) - (CELL_LENGTH_X * CELL_CUTOFF_RADIUS) * floor(cell.particle_list[threadIdx.x].x / (CELL_LENGTH_X * CELL_CUTOFF_RADIUS));
    cell.particle_list[threadIdx.x].y = (cell.particle_list[threadIdx.x].y + cell.particle_list[threadIdx.x].vy * TIMESTEP_DURATION) - (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS) * floor(cell.particle_list[threadIdx.x].y / (CELL_LENGTH_Y * CELL_CUTOFF_RADIUS));
    cell.particle_list[threadIdx.x].z = (cell.particle_list[threadIdx.x].z + cell.particle_list[threadIdx.x].vz * TIMESTEP_DURATION) - (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS) * floor(cell.particle_list[threadIdx.x].z / (CELL_LENGTH_Z * CELL_CUTOFF_RADIUS));

    cell_list[cell_x + cell_y * CELL_LENGTH_X + cell_z * CELL_LENGTH_X * CELL_LENGTH_Y].particle_list[threadIdx.x] = cell.particle_list[threadIdx.x];

    // put above code in force_eval. this code is supposed to 
    // update cell list with updated particles
    // one block per cell
    // one thread per cell not equal to block's cell
    // that thread loops over each particle and copies it to home cell's particle list

    accelerations[particleId] = 0;
    accelerations[particleId + 1] = 0;
    accelerations[particleId + 2] = 0;
}

void initialize_cell_list(struct Cell cellList[CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z])
{
        // initialize cell list, -1 for empty cell
        memset(cellList, -1, sizeof(cellList));
        for (int i = 0; i < NUM_PARTICLES; ++i) {
                int x = rand() % CELL_LENGTH_X;
                int y = rand() % CELL_LENGTH_Y;
                int z = rand() % CELL_LENGTH_Z;

                struct Particle particle = {
                        .particleId = i,
                        .x = x * CELL_CUTOFF_RADIUS + ((float) rand() / RAND_MAX) * CELL_CUTOFF_RADIUS,
                        .y = y * CELL_CUTOFF_RADIUS + ((float) rand() / RAND_MAX) * CELL_CUTOFF_RADIUS,
                        .z = z * CELL_CUTOFF_RADIUS + ((float) rand() / RAND_MAX) * CELL_CUTOFF_RADIUS,
                        .vx = 0,
                        .vy = 0,
                        .vz = 0,
                };
                // copy particle to 
                for (int j = 0; j < MAX_PARTICLES_PER_CELL; ++j) {
                    if (cellList[x][y][z].particle_list[j].particleId == -1) {
                        memcpy(&cellList[x][y][z].particle_list[j], &particle, sizeof(struct Particle));
                        break;
                    }
                }
        }
}

int main() 
{
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // INITIALIZE CELL LIST WITH PARTICLE DATA
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // initialize (or import) particle data for simulation
    struct Cell cell_list[CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z];
    initialize_cell_list(&cell_list);
    // device_cell_list stores an array of Cells, where each Cell contains a particle_list
    struct Cell *device_cell_list;
    // hipMalloc initializes GPU global memory to be used as parameter for GPU kernel
    hipMalloc(&device_cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell));
    hipMemcpy(device_cell_list, cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * sizeof(struct Cell), hipMemcpyHostToDevice);


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // INITIALIZE ACCELERATIONS
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /*
        accelerations stores accelerations (in x y z dimensions) of each particle to be used in motion update.
        - index of accelerations is related to particleId
        - particleId * 3 gives index of accelerations for x dimension
        - (particleId * 3) + 1 gives index of y
        - (particleId * 3) + 2 gives index of y
    */
    float *accelerations;
    hipMalloc(&accelerations, MAX_PARTICLES_PER_CELL * 3 * sizeof(float));
    hipMemset(accelerations, 0, MAX_PARTICLES_PER_CELL * 3 * sizeof(float));


    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // INITIALIZE PARAMETERS FOR FORCE COMPUTATION AND MOTION UPDATE
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // defines block and thread dimensions
    // dim3 is an integer vector type most commonly used to pass the grid and block dimensions in a kernel invocation [X x Y x Z]
    dim3 numBlocksForce(CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * 14);    // (CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * 14) x 1 x 1
    dim3 numBlocksMotion(CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z);        // (CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z) x 1 x 1
    dim3 threadsPerBlock(MAX_PARTICLES_PER_CELL);                               // MAX_PARTICLES_PER_CELL x 1 x 1

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // FORCE COMPUTATION AND MOTION UPDATE
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // do force evaluation and motion update for each time step
    // steps are separated to ensure threads are synchronized (that force_eval is done)
    // output of force_eval is stores in device_cell_list and accelerations
    for (int t = 0; t < TIMESTEPS; ++t) {
        force_eval<<<numBlocksForce, threadsPerBlock>>>(device_cell_list, accelerations);
        motion_update<<<numBlocksMotion, threadsPerBlock>>>(device_cell_list, accelerations);
    }

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //  COPY FINAL RESULT BACK TO HOST CPU
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    hipMemcpy(cell_list, device_cell_list, CELL_LENGTH_X * CELL_LENGTH_Y * CELL_LENGTH_Z * struct(struct Cell), hipMemcpyDeviceToHost);
    hipFree(device_cell_list);

    return 0;
}
